#include <chrono>
#include <hip/hip_runtime.h>

#include <iostream>

#define N 10000000
#define MAX_ERR 1e-6

/**
Compile with
LD_LIBRARY_PATH=/usr/local/cuda/compat nvcc vector_add.cu -Xcompiler -O3 \
-arch=sm_86 -o vector_add
 */

void vector_add_cpu(float *out, float *a, float *b, int n) {
  for (size_t i = 0; i < n; i++)
    out[i] = a[i] + b[i];
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < n) {
    out[idx] = a[idx] + b[idx];
  }
}

int main() {
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;

  a = (float *)malloc(sizeof(float) * N);
  b = (float *)malloc(sizeof(float) * N);
  out = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  hipError_t err = hipMalloc((void **)&d_a, sizeof(float) * N);
  err = hipMalloc((void **)&d_b, sizeof(float) * N);
  err = hipMalloc((void **)&d_out, sizeof(float) * N);
  err = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  err = hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  vector_add<<<numBlocks, blockSize>>>(d_out, d_a, d_b, N);
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();

  std::cout << "GPU time: " << std::chrono::duration<float>(end - start).count()
            << "\n";

  err = hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  start = std::chrono::high_resolution_clock::now();
  vector_add_cpu(out, a, b, N);
  end = std::chrono::high_resolution_clock::now();

  std::cout << "CPU time: " << std::chrono::duration<float>(end - start).count()
            << "\n";

  free(a);
  free(b);
  free(out);
}
