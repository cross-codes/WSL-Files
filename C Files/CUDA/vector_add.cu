#include <assert.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < n) {
    out[idx] = a[idx] + b[idx];
  }
}

int main() {
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;

  a = (float *)malloc(sizeof(float) * N);
  b = (float *)malloc(sizeof(float) * N);
  out = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  hipError_t err = hipMalloc((void **)&d_a, sizeof(float) * N);
  if (err != hipSuccess) {
    printf("CUDA malloc failed for d_a: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipMalloc((void **)&d_b, sizeof(float) * N);
  if (err != hipSuccess) {
    printf("CUDA malloc failed for d_b: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipMalloc((void **)&d_out, sizeof(float) * N);
  if (err != hipSuccess) {
    printf("CUDA malloc failed for d_out: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA memcpy failed for d_a: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("CUDA memcpy failed for d_b: %s\n", hipGetErrorString(err));
    return -1;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  vector_add<<<numBlocks, blockSize>>>(d_out, d_a, d_b, N);

  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA synchronization failed: %s\n", hipGetErrorString(err));
    return -1;
  }

  err = hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("CUDA memcpy failed for d_out: %s\n", hipGetErrorString(err));
    return -1;
  }

  for (int i = 0; i < 10; i++) {
    printf("out[%d] = %f, expected = %f\n", i, out[i], a[i] + b[i]);
  }

  for (int i = 0; i < N; i++) {
    if (fabs(out[i] - (a[i] + b[i])) > MAX_ERR) {
      printf("Error at index %d: Expected %f, but got %f\n", i, a[i] + b[i],
             out[i]);
      return 1;
    }
  }

  printf("PASSED\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  free(a);
  free(b);
  free(out);
}
